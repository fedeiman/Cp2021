#include "test.cu"

char t1[] = "Tiny Monte Carlo by Scott Prahl (http://omlc.ogi.edu)";
char t2[] = "1 W Point Source Heating in Infinite Isotropic Scattering Medium";
char t3[] = "CPU version, adapted for PEAGPGPU by Gustavo Castellano"
            " and Nicolas Wolovick";
char error[] = "Error";



int main()
{

    if (verbose) { // heading
        printf("# %s\n# %s\n# %s\n", t1, t2, t3);
        printf("# Scattering = %8.3f/cm\n", MU_S);
        printf("# Absorption = %8.3f/cm\n", MU_A);
        printf("# Photons    = %8d\n#\n", PHOTONS);
    }

    float ** heat;

    if (hipMallocManaged(&heat, SHELLS * sizeof(float *)) != hipSuccess) return printf("# %s\n#", error);
    for (int i = 0; i < SHELLS; i++) {
        if(hipMallocManaged(&heat[i], 2 * sizeof(float))!= hipSuccess ) return printf("# %s\n#", error);
    }

    (void)run_gpu_tiny_mc(heat, PHOTONS);

    for (int i = 0; i < SHELLS; i++) {
        heat[i][0] = heat[i][1] = 0.0f;
    }


    double start = wtime();
    (void)run_gpu_tiny_mc(heat, PHOTONS);
    double end = wtime();
    assert(start <= end);
    double elapsed = (end - start);

    printf("%lf\n", 1e-3 * PHOTONS / elapsed);



    return 0;
}