#include "hip/hip_runtime.h"
/* Tiny Monte Carlo by Scott Prahl (http://omlc.ogi.edu)"
 * 1 W Point Source Heating in Infinite Isotropic Scattering Medium
 * http://omlc.ogi.edu/software/mc/tiny_mc.c
 *
 * Adaptado para CP2014, Nicolas Wolovick
 */

#define _XOPEN_SOURCE 500  // M_PI

#include "params.h"
#include "wtime.h"

#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    hipDeviceReset(); \
    return EXIT_FAILURE;}} while(0)

char t1[] = "Tiny Monte Carlo by Scott Prahl (http://omlc.ogi.edu)";
char t2[] = "1 W Point Source Heating in Infinite Isotropic Scattering Medium";
char t3[] = "CPU version, adapted for PEAGPGPU by Gustavo Castellano"
            " and Nicolas Wolovick";

typedef struct PhotonHeat {
    float *heat;
    float *heat2;
} PhotonHeat;

__global__ void setup_kernel(hiprandState *state, uint64_t seed)
{
    int gtid = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(seed, gtid, 0, &state[gtid]);
}
/***
    Each Photon in a thread with its own RNG
 ***/
static __global__ void photon(PhotonHeat result, hiprandState *rngs)
{
    int gtid = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Most of the time we'll have more threads than PHOTONS due to ceil()
    // In block calculations, only do required photons
    if(gtid <= PHOTONS){
        const float albedo = MU_S / (MU_S + MU_A);
        const float shells_per_mfp = 1e4 / MICRONS_PER_SHELL / (MU_A + MU_S);

        /* launch */
        float x = 0.0f;
        float y = 0.0f;
        float z = 0.0f;
        float u = 0.0f;
        float v = 0.0f;
        float w = 1.0f;
        float weight = 1.0f;
        int prevShell = -1;
        float cachedW= 0;
        float cachedWS= 0;

        for (;;) {
            float t = -logf(hiprand_uniform(&rngs[gtid])); /* move */
            x += t * u;
            y += t * v;
            z += t * w;

            int shell = sqrtf(x * x + y * y + z * z) * shells_per_mfp; /* absorb */
            if (shell > SHELLS - 1) {
                shell = SHELLS - 1;
            }
            if(prevShell == shell){
                cachedW += (1.0f - albedo) * weight;
                cachedWS += (1.0f - albedo) * (1.0f - albedo) * weight * weight;
            }
            else{
              //drop cached value in global heat array 
              if(prevShell >= 0){
                atomicAdd(&result.heat[prevShell], cachedW);
                atomicAdd(&result.heat2[prevShell], cachedWS); 
              }
              cachedW = (1.0f - albedo) * weight;
              cachedWS = (1.0f - albedo) * (1.0f - albedo) * weight * weight;
              prevShell = shell;
            }

            weight *= albedo;
            if (weight < 0.001f) { /* roulette */
                //If we break we must put the cached value in the array
                if (hiprand_uniform(&rngs[gtid]) > 0.1f){
                    atomicAdd(&result.heat[prevShell], cachedW);
                    atomicAdd(&result.heat2[prevShell], cachedWS); 
                    break;
                }
                    
                weight *= 10;
            }
            /* New direction, rejection method */
            float xi1, xi2;
            do {
                xi1 = 2.0f * hiprand_uniform(&rngs[gtid]) - 1.0f;
                xi2 = 2.0f * hiprand_uniform(&rngs[gtid]) - 1.0f;
                t = xi1 * xi1 + xi2 * xi2;
            } while (1.0f < t);
            u = 2.0f * t - 1.0f;

            float tmp = sqrtf((1.0f - u * u) / t);
            v = xi1 * tmp;
            w = xi2 * tmp;
        }
    }
}


/***
 * Main matter
 ***/

int main(void)
{

    // heading
    /* printf("# %s\n# %s\n# %s\n", t1, t2, t3);
    printf("# Scattering = %8.3f/cm\n", MU_S);
    printf("# Absorption = %8.3f/cm\n", MU_A);
    printf("# Photons    = %8d\n#\n", PHOTONS); */
    float blocks = ceil(PHOTONS / 128);
    int num_threads = blocks * 128;

    hiprandState *rngs;
    CUDA_CALL(hipMallocManaged(&rngs, num_threads * sizeof(hiprandState)));

    PhotonHeat global;
    CUDA_CALL(hipMallocManaged(&global.heat, SHELLS * sizeof(float)));
    CUDA_CALL(hipMallocManaged(&global.heat2, SHELLS * sizeof(float)));
    CUDA_CALL(hipMemset (global.heat,0, SHELLS));
    CUDA_CALL(hipMemset (global.heat2,0, SHELLS));

    //Initialize rngs, same seed, different sequence
    setup_kernel << < blocks, 128 >> > (rngs, time(NULL));

    hipEvent_t start, stop;
    // start timer
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventRecord(start,0));
    
    // simulation
    photon<<<blocks, 128>>>(global, rngs);

    // stop timer
    CUDA_CALL(hipEventCreate(&stop));
    CUDA_CALL(hipEventRecord(stop,0));
    CUDA_CALL(hipEventSynchronize(stop));
    float elapsed;
    CUDA_CALL(hipEventElapsedTime(&elapsed, start,stop));
    elapsed /= 1000;

    // printf("# %lf seconds\n", elapsed);
    //printf("# %lf K photons per second\n", 1e-3 * PHOTONS / elapsed);
    printf("%lf\n", 1e-3 * PHOTONS / elapsed);
     /* printf("# Radius\tHeat\n");
    printf("# [microns]\t[W/cm^3]\tError\n");
    float t = 4.0f * M_PI * powf(MICRONS_PER_SHELL, 3.0f) * PHOTONS / 1e12;
    for (unsigned int i = 0; i < SHELLS - 1; ++i) {
        printf("%6.0f\t%12.5f\t%12.5f\n", i * (float)MICRONS_PER_SHELL,
               global.heat[i] / t / (i * i + i + 1.0 / 3.0),
               sqrt(global.heat2[i] - global.heat[i] * global.heat[i] / PHOTONS) / t / (i * i + i + 1.0f / 3.0f));
    }
    printf("# extra\t%12.5f\n", global.heat[SHELLS - 1] / PHOTONS);  */

    CUDA_CALL(hipFree(global.heat));
    CUDA_CALL(hipFree(global.heat2));
    CUDA_CALL(hipFree(rngs));
    return 0;
}
